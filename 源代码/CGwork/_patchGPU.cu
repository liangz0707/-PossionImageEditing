#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include<hiprand/hiprand_kernel.h>
//�������
__device__ float calcDistance(int target_row ,int target_col,int source_row, int source_col , double * target_block ,double * source_block,int target_rows,int target_cols ,int source_rows,int source_cols)
{	
	int r = 3;
	double dif=0;
	double dif0=1,dif1=1,dif2=1;

	if( target_row-r<0 )  target_row = r;
	if( target_col-r<0 )  target_col =r;
	if( source_row-r<0 )  source_row = r;
	if( source_col-r<0 )  source_col = r;

	if( target_row+r>=target_rows )  target_row = target_cols-1-r;
	if( target_col+r>= target_cols )  target_col = target_rows-1-r;
	if( source_row+r>=source_rows  )  source_row = source_rows-1-r;
	if( source_col+r>= source_cols )  source_col = source_cols-1-r;	

	for(int i=-r ;i<=r;i++){
		for(int j=-r ;j<=r;j++){
			int temp = 3*((source_row+i)*source_cols+source_col+j) ;
			int temp2 = 3*((target_row+i)*target_cols+target_col+j) ;
			dif0 = source_block[ temp+ 0] - target_block[temp2+ 0] ;
			dif1 = source_block[ temp+ 1] - target_block[temp2+ 1] ;
			dif2 = source_block[ temp+ 2] - target_block[temp2+ 2] ;
			dif += sqrt(dif0*dif0 +dif1*dif1 +dif2*dif2);
		}	
	}
	return dif;
}
__device__ int calcDistance(int target_row ,int target_col,int source_row1, int source_col1,int source_row2, int source_col2 ,int source_row3, int source_col3,double * target_block , double *source_block,int target_rows,int target_cols ,int source_rows,int source_cols)
{
	float first2Second = calcDistance(target_row,  target_col , source_row1,  source_col1,target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	float first2Third = calcDistance(  target_row, target_col , source_row2,  source_col2,target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	float first2Fourth = calcDistance( target_row,target_col , source_row3,  source_col3,target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	if (first2Second<=first2Third)
	{
		if (first2Second<=first2Fourth)
			return 1;
		else
			return 3;
	}
	else if (first2Third<= first2Fourth)
		return 2;
	else
		return 3;
}
__device__ int calcDistance(int target_row,int target_col ,int source_row1, int source_col1,int source_row2, int source_col2 ,double * target_block ,double * source_block,int target_rows,int target_cols ,int source_rows,int source_cols)
{
	float first2Second = calcDistance(target_row,  target_col , source_row1,  source_col1,target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	float first2Third = calcDistance(  target_row, target_col , source_row2,  source_col2,target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	if (first2Second <= first2Third)
		return 1;
	return 2;
}

__global__ void extern PropagationGPU(double * target_block ,double * source_block ,  int * relation_block  ,  int target_rows  ,  int target_cols ,int source_rows  ,  int source_cols)
{
	//���ݹ��̣�
	int y = blockIdx.x;
	int x = threadIdx.x;

	//����뾶��
	int c_r0 = relation_block[ 2*(y*target_cols+x) + 0 ];
	int c_c0 = relation_block[ 2*(y*target_cols+x) + 1];

	int c_r1 = relation_block[ 2*((y+1)*target_cols+x) + 0 ]-1;
	int c_c1 = relation_block[ 2*((y+1)*target_cols+x) + 1 ];

	int c_r2 = relation_block[ 2*(y*target_cols+x+1) + 0];
	int c_c2 = relation_block[ 2*(y*target_cols+x+1) + 1]-1;

	int patchNumber = calcDistance(y  ,  x  ,  c_r0 ,  c_c0 , c_r1, c_c1 , c_r2 , c_c2 , target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	switch(patchNumber)
	{ 
		case 2:
			relation_block[ 2*(y*target_cols+x) + 0 ]= c_r1;
			relation_block[ 2*(y*target_cols+x) + 1 ]= c_c1;
			break;
		case 3:
			relation_block[ 2*(y*target_cols+x) + 0 ] = c_r2;
			relation_block[ 2*(y*target_cols+x) + 1 ] = c_c2;
			break;
	}
}
__global__ void extern RandomSearchGPU(double * target_block ,double * source_block ,int * relation_block,int target_rows,int target_cols ,int source_rows,int source_cols){
	//���ݹ��̣�
	int y = blockIdx.x;
	int x = threadIdx.x;

	//����뾶��
	int c_r0 = relation_block[ 2*(y*target_cols+x) + 0 ];
	int c_c0 = relation_block[ 2*(y*target_cols+x) + 1];

	int c_r1 = relation_block[ 2*((y-2)*target_cols+x) + 0 ]+2;
	int c_c1 = relation_block[ 2*((y-2)*target_cols+x) + 1 ];

	int c_r2 = relation_block[ 2*(y*target_cols+x-2) + 0];
	int c_c2 = relation_block[ 2*(y*target_cols+x-2) + 1]+2;

	int patchNumber = calcDistance(y  ,  x  ,  c_r0 ,  c_c0 , c_r1, c_c1 , c_r2 , c_c2 , target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
	switch(patchNumber)
	{
		case 2:
			relation_block[ 2*(y*target_cols+x) + 0 ]= c_r1;
			relation_block[ 2*(y*target_cols+x) + 1 ]= c_c1;
			break;
		case 3:
			relation_block[ 2*(y*target_cols+x) + 0 ] = c_r2;
			relation_block[ 2*(y*target_cols+x) + 1 ] = c_c2;
			break;
	}
}

__global__ void extern baoli(double * target_block ,double * source_block ,int * relation_block,int target_rows,int target_cols ,int source_rows,int source_cols,double *distance){
	int y = threadIdx.y;
	int x = threadIdx.x;

	for(int i  = 0 ; i<12;i++){
		for(int j =  0 ;j< 12 ; j++){
			double c =  calcDistance(y  ,  x  ,  i ,  j ,  target_block ,source_block,target_rows,target_cols,source_rows, source_cols);
			if( c < distance[ y*target_cols+x ]){
				relation_block[ 2*(y*target_cols+x) + 0 ]= 1;
				relation_block[ 2*(y*target_cols+x) + 1 ]= 1;
				distance[ y*target_cols+x ] = c;
			}
		}
	}
}

void extern  bridge(double * target_block ,double * source_block  ,int *  relation_block,int target_rows,int target_cols ,int source_rows,int source_cols, double * distance){
	//���̴߳�С�����ƣ�
/**/	for(int i = 0;i<130 ;i++){
		PropagationGPU<<<target_rows  ,target_cols>>>(target_block, source_block, relation_block , target_rows , target_cols , source_rows , source_cols);
		hipDeviceSynchronize();
		RandomSearchGPU<<<target_rows	,target_cols>>>(target_block, source_block, relation_block,target_rows,target_cols,source_rows, source_cols);
		hipDeviceSynchronize();
	}

	//baoli<<<target_rows  ,target_cols>>>(target_block, source_block, relation_block , target_rows , target_cols , source_rows , source_cols	,distance);
}